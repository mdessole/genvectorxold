#include "hip/hip_runtime.h"
#include "SYCLMath/Vector4D.h"
#include "SYCLMath/VecOps.h"
#include <chrono>
#include <vector>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>


using arithmetic_type = double;
using vec4d = ROOT::Experimental::LorentzVector<
    ROOT::Experimental::PtEtaPhiM4D<arithmetic_type>>;
template <class T>
using Vector = std::vector<T>;



vec4d* GenVectors(int n)
{

  vec4d* vectors = new vec4d[n];

  // generate n -4 momentum quantities
  for (int i = 0; i < n; ++i)
  {
    // fill vectors
    vectors[i] = {1., 1., 1., 1.};
  }

  return vectors;
}

bool print_if_false(const bool assertion, size_t i) {
  if (!assertion) {
    std::cout << "Assertion failed at index "<< i << std::endl;
  }
  return assertion;
}

int main(int argc, char **argv)
{

  std::string arg1 = argv[1];
  std::size_t pos;
  std::size_t N = std::stoi(arg1, &pos);
  size_t local_size = 128;

  vec4d* u_vectors = GenVectors(N);
  vec4d* v_vectors = GenVectors(N);


  arithmetic_type* masses = ROOT::Experimental::InvariantMasses<arithmetic_type, vec4d>(u_vectors, v_vectors, N, local_size);

  for (size_t i=0; i<N; i++)
    assert(print_if_false((std::abs(masses[i] - 2.) <= 1e-5), i) );

  delete[] masses;
  return 0;
}

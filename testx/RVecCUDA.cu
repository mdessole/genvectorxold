#include "hip/hip_runtime.h"
#include "SYCLMath/Vector4D.h"
#include <chrono>
#include <vector>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>


using arithmetic_type = double;
using vec4d = ROOT::Experimental::LorentzVector<
    ROOT::Experimental::PtEtaPhiM4D<arithmetic_type>>;
template <class T>
using Vector = std::vector<T>;

#ifndef RVecCUDA_H
#define RVecCUDA_H

// namespace ROOT {
// namespace Experimental {

template <class Vec, class Mass>
__global__ void InvariantMassKernel(Vec vec, Mass m, size_t N)
{
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < N)
  {
    vec4d w = vec[id];
    m[id] = w.mass();
  }
}

template <class Vec, class Mass>
__global__ void InvariantMassesKernel(Vec v1, Vec v2, Mass m, size_t N)
{
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < N)
  {
    vec4d w = v1[id];
    w+=v2[id];
    m[id] = w.mass();
  }
}

//} // namespace Experimental
//} // namespace ROOT
#endif

arithmetic_type* InvariantMasses(vec4d* v1, vec4d* v2,  const size_t N,
                                      const size_t local_size)
{

  arithmetic_type* invMasses = new arithmetic_type[N];

  size_t sizeVec = N*sizeof(vec4d);
  auto start = std::chrono::system_clock::now();
  hipError_t err;
{
// Allocate the device input vector A
  vec4d* d_v1 = NULL;
  err = hipMalloc((void **)&d_v1, sizeVec);


  // Allocate the device input vector B
  vec4d* d_v2 = NULL;
  err = hipMalloc((void **)&d_v2, sizeVec);

  // Allocate the device output vector C
  arithmetic_type* d_invMasses = NULL;
  err = hipMalloc((void **)&d_invMasses, sizeVec);

  hipMemcpy ( d_v1, v1, sizeVec, hipMemcpyHostToDevice );
  hipMemcpy ( d_v2, v2, sizeVec, hipMemcpyHostToDevice );

  InvariantMassesKernel<<<fmax(1, N / local_size), local_size>>>(d_v1, d_v2, d_invMasses, N);

  hipMemcpy ( invMasses, d_invMasses, sizeVec, hipMemcpyDeviceToHost );
 

  hipFree(d_v1);
  hipFree(d_v2);
  hipFree(d_invMasses);

}
 
  auto end = std::chrono::system_clock::now();
  auto duration =
      std::chrono::duration_cast<std::chrono::microseconds>(end - start)
          .count() *
      1e-6;
  std::cout << "cuda time " << duration << " (s)" << std::endl;


  return invMasses;
}

arithmetic_type* InvariantMass(vec4d* v1, const size_t N, const size_t local_size)
{

  arithmetic_type* invMasses = new arithmetic_type[N];

  size_t sizeVec = N*sizeof(vec4d);
  auto start = std::chrono::system_clock::now();
  hipError_t err;
{
// Allocate the device input vector A
  vec4d* d_v1 = NULL;
  err = hipMalloc((void **)&d_v1, sizeVec);



  // Allocate the device output vector C
  arithmetic_type* d_invMasses = NULL;
  err = hipMalloc((void **)&d_invMasses, sizeVec);

  hipMemcpy ( d_v1, v1, sizeVec, hipMemcpyHostToDevice );


  InvariantMassKernel<<<fmax(1, N / local_size), local_size>>>(d_v1, d_invMasses, N);

  hipMemcpy ( invMasses, d_invMasses, sizeVec, hipMemcpyDeviceToHost );
 
  hipFree(d_v1);
  hipFree(d_invMasses);

}
 
  auto end = std::chrono::system_clock::now();
  auto duration =
      std::chrono::duration_cast<std::chrono::microseconds>(end - start)
          .count() *
      1e-6;
  std::cout << "cuda time " << duration << " (s)" << std::endl;


  return invMasses;
}

vec4d* GenVectors(int n)
{

  vec4d* vectors = new vec4d[n];

  // generate n -4 momentum quantities
  for (int i = 0; i < n; ++i)
  {
    // fill vectors
    vectors[i] = {1., 1., 1., 1.};
  }

  return vectors;
}

int main(int argc, char **argv)
{

  std::string arg1 = argv[1];
  std::size_t pos;
  std::size_t N = std::stoi(arg1, &pos);
  size_t local_size = 128;

  vec4d* u_vectors = GenVectors(N);
  vec4d* v_vectors = GenVectors(N);


  arithmetic_type* masses = InvariantMasses(u_vectors, v_vectors, N, local_size);

//InvariantMass(u_vectors, N, local_size);
  //    InvariantMasses(u_vectors, v_vectors, N, local_size);

  //std::cout << "masses[0] " << masses[5] << std::endl;
  assert((std::abs(masses[0] - 2.) <= 1e-5));
  return 0;
}
